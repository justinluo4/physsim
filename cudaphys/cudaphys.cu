#include "hip/hip_runtime.h"
﻿// physsim.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include<math.h>
#include<chrono>
#include<thread>
#include <future>
#include <string>
#define SDL_MAIN_HANDLED
#include<SDL2/SDL.h>
#include <sstream>
#include "cdraw.h"
#include "vec.h"

using namespace v;
using namespace draw;

bool quit = false;
int grabbed = -1;

struct PhysObj {
	double mass, rv, radius;
	vec2 vel, pos;
	bool do_gravity, do_collide;
};


auto start = std::chrono::system_clock::now();
auto end = std::chrono::system_clock::now();
vec2 mousepos = { 0, 0 };
vec2 lastmousepos;
vec2 graboffset;
std::stringstream ss;
const double G = 10;
const double CR = 0;
const double FRIC = 1;
const double dt = 0.00001;
int frame_tick = 0;
int tpf = 1;
int N = 1000;
PhysObj* objects;
PhysObj* dev_objects;
PhysObj* dev_objects_write;
__device__
inline vec2 devMul(vec2 v1, double k) {
	return { v1.x * k , v1.y * k };
}
__device__
inline vec2 devAdd(vec2 v1, vec2 v2) {
	return { v1.x + v2.x , v1.y + v2.y };
}
__device__
inline vec2 devSub(vec2 v1, vec2 v2) {
	return { v1.x - v2.x , v1.y - v2.y };
}
__device__
inline double devLen(vec2 v1) {
	return powf(pow(v1.x, 2) + pow(v1.y, 2), 0.5);
}
__device__
inline double devDist(vec2 v1, vec2 v2) {
	return powf(pow(v1.x - v2.x, 2) + pow(v1.y - v2.y, 2), 0.5);
} 
__device__
inline vec2 devNormalize(vec2 v1, vec2 v2) {
	vec2 sub = devSub(v2 , v1);
	sub = devMul(sub, (1.0 / devLen(sub)));
	return sub;
}
__device__
inline double devDot(vec2 v1, vec2 v2) {
	return v1.x * v2.x + v1.y * v2.y;
}
__global__
void collide(int N, PhysObj* objs, PhysObj* objs_write, double CR, double FRIC)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;
	if (i < N && j < N) {
		PhysObj& obj1 = objs[i];
		PhysObj& obj2 = objs[j];
		double dist = devDist(obj1.pos, obj2.pos);
		if (i < j && objs[i].do_collide && objs[j].do_collide && dist <= obj1.radius + obj2.radius) {
			vec2 dir = devNormalize(obj1.pos, obj2.pos);
			vec2 tan = { -dir.y, dir.x };

			double overlap = obj1.radius + obj2.radius - dist;


			double pv1 = devDot(dir, obj1.vel);
			double pv2 = devDot(dir, obj2.vel);
			double cv1 = FRIC * devDot(tan, obj1.vel);
			double cv2 = FRIC * devDot(tan, obj2.vel);
			double dv1 = (CR * obj2.mass * (pv2 - pv1) + obj1.mass * pv1 + obj2.mass * pv2) / (obj1.mass + obj2.mass);
			double dv2 = (CR * obj1.mass * (pv1 - pv2) + obj1.mass * pv1 + obj2.mass * pv2) / (obj1.mass + obj2.mass);
			vec2 diff = devAdd(devMul(dir, dv1), devMul(tan, cv1));
			atomicAdd(&objs_write[i].vel.x, diff.x -obj1.vel.x);
			atomicAdd(&objs_write[i].vel.y, diff.y -obj1.vel.y);
			diff = devAdd(devMul(dir, dv2), devMul(tan, cv2));
			atomicAdd(&objs_write[j].vel.x, diff.x -obj2.vel.x);
			atomicAdd(&objs_write[j].vel.y, diff.y -obj2.vel.y);
			diff = devMul(dir, overlap * (obj2.mass / (obj1.mass + obj2.mass)) * -1);
			atomicAdd(&objs_write[i].pos.x, diff.x);
			atomicAdd(&objs_write[i].pos.y, diff.y);
			diff = devMul(dir, overlap * (obj1.mass / (obj1.mass + obj2.mass)));
			atomicAdd(&objs_write[j].pos.x, diff.x);
			atomicAdd(&objs_write[j].pos.y, diff.y);
		}
	}

}
__global__
void gravitate(int N, PhysObj* objs, PhysObj* objs_write, double G, double dt) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = index / N;
	int j = index % N;
	if (i < N && j < N) {
		PhysObj& obj1 = objs[i];
		PhysObj& obj2 = objs[j];
		if (i != j && objs[i].do_gravity && objs[j].do_gravity) {
			vec2 diff = devMul(devNormalize(obj1.pos, obj2.pos), G * dt * obj2.mass / devDot(devSub(obj2.pos, obj1.pos), devSub(obj2.pos, obj1.pos)));
			atomicAdd(&objs_write[i].vel.x, diff.x);
			atomicAdd(&objs_write[i].vel.y, diff.y);
		}
	}
}
__global__
void step(int N,  PhysObj* objs, PhysObj* objs_write, double dt) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N){
		PhysObj& obj = objs[i];
		vec2 diff = devMul(obj.vel, dt);
		atomicAdd(&objs_write[i].pos.x, diff.x);
		atomicAdd(&objs_write[i].pos.y, diff.y);
	}

}

void main_loop()
{
	//Handle events on queue
	while (SDL_PollEvent(&e) != 0)
	{
		switch (e.type)
		{
		case SDL_QUIT:
			quit = true;
			break;
		
		case SDL_MOUSEBUTTONDOWN:
			switch (e.button.button)
			{
			case SDL_BUTTON_LEFT:
				mousepos = { (float)e.motion.x / 100, (float)e.motion.y / 100 };
				ss << "X: " << mousepos.x << " Y: " << mousepos.y;

				SDL_SetWindowTitle(gWindow, ss.str().c_str());
				for (int i = 0; i < N; i++) {
					if (dist(mousepos, objects[i].pos) <= objects[i].radius) {
						grabbed = i;
						graboffset = objects[i].pos - mousepos;
					}
				}
				break;
			case SDL_BUTTON_RIGHT:
				grabbed = -1;
				break;
		
			}
			
		}

	}
	int mouseX, mouseY;
	SDL_GetMouseState(&mouseX, &mouseY);
	start = end;
	end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	lastmousepos = mousepos;
	mousepos = { (float)mouseX / 100, (float)mouseY / 100 };
	if (grabbed != -1) {
		objects[grabbed].pos = mousepos + graboffset;
		objects[grabbed].vel = (mousepos - lastmousepos)*(1/ elapsed_seconds.count());
	}
	//Clear screen
	SDL_SetRenderDrawColor(gRenderer, 0x0, 0x0, 0x0, 0xFF);
	SDL_RenderClear(gRenderer);
	for (int i = 0; i < N; i++) {
		PhysObj& obj = objects[i];
		fill_circle(gRenderer, 100 * obj.pos.x, 100 * obj.pos.y, obj.radius * 100, 0x00, obj.mass / 10, 0xFF, 0xFF);
	}



	//Update screen
	SDL_RenderPresent(gRenderer);
}
int main(int argc, char* args[])
{
	size_t size = N * sizeof(PhysObj);
	objects = (PhysObj*)malloc(size);
	hipMalloc((void**)&dev_objects, size);
	hipMalloc((void**)&dev_objects_write, size);
	int blockSize;
	int numBlocks;
	SDL_SetMainReady();
	//Start up SDL and create window
	srand(time(NULL));
	/*
	for(int c = 0; c < 40; c++) {
		double osize = (float)rand() / RAND_MAX * 0.3 + 0.03;
		objects.push_back({ osize*1000, 0.0, osize, {(float)rand() / RAND_MAX * 10 - 5, (float)rand() / RAND_MAX * 10 - 5}, {(float)rand() / RAND_MAX * 10, (float)rand() / RAND_MAX * 5} });
	}*/
	int rows = 50;
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < N / rows; j++) {
			objects[i+rows*j] = { (double)5, 0, 0.03, {100, 0}, {3+ j * 0.06 , 5 + i*0.06} , true, true};
		}

	}
	objects[0] = { (double)10000, 0, 0.15, {0, 0}, {7.5123 , 2.6} , true, true};
	/*
	for (int j = 0; j < N ; j++) {
		objects[j] = { (double)100, 0, 0.1, {0, 0}, {4 + j * 0.21, 2.0 } };
	}
	*/
	/*
	for (int p = 0; p < 30; p++) {
		objects.push_back({ 50, 0, 0.04, {10, 0}, {p * 0.1, 2.5} });
	}
	for (int p = 0; p < 30; p++) {
		objects.push_back({ 50, 0, 0.04, {-10, 0}, {p * 0.1, 3.5} });
	}
	*/
	if (!init())
	{
		printf("Failed to initialize!\n");
	}
	else
	{
		//Load media
		if (!loadMedia())
		{
			printf("Failed to load media!\n");
		}
		else
		{
#ifdef __EMSCRIPTEN__
			emscripten_set_main_loop(main_loop, 0, 1);
#else

			//While application is running
			while (!quit)
			{
				//Host Collide
				/*
				for (int i = 0; i < N; i++) {
					for (int j = 0; j < N; j++) {
						PhysObj& obj1 = objects[i];
						PhysObj& obj2 = objects[j];
						if ((obj1.pos.x != obj2.pos.x || obj1.pos.y != obj2.pos.y) && dist(obj1.pos, obj2.pos) < obj1.radius + obj2.radius) {
							vec2 dir = normalize(obj1.pos, obj2.pos);
							vec2 tan = { -dir.y, dir.x };

							double overlap = obj1.radius + obj2.radius - dist(obj1.pos, obj2.pos);
							obj1.pos += dir * overlap * (obj2.mass / (obj1.mass + obj2.mass)) * -1;
							obj2.pos += dir * overlap * (obj1.mass / (obj1.mass + obj2.mass));

							double pv1 = dot(dir, obj1.vel);
							double pv2 = dot(dir, obj2.vel);
							double cv1 = dot(tan, obj1.vel);
							double cv2 = dot(tan, obj2.vel);
							double dv1 = (CR * obj2.mass * (pv2 - pv1) + obj1.mass * pv1 + obj2.mass * pv2) / (obj1.mass + obj2.mass);
							double dv2 = (CR * obj1.mass * (pv1 - pv2) + obj1.mass * pv1 + obj2.mass * pv2) / (obj1.mass + obj2.mass);
							obj1.vel = dir * dv1 + tan * cv1;
							obj2.vel = dir * dv2 + tan * cv2;
						}

					}
				}
				*/

				// GPU Section
				blockSize = 256;
				numBlocks = (N*N + blockSize - 1)/blockSize;
				hipMemcpy(dev_objects, objects, size, hipMemcpyHostToDevice);
				hipMemcpy(dev_objects_write, dev_objects, size, hipMemcpyDeviceToDevice);

				collide << <numBlocks, blockSize >> > (N, dev_objects,dev_objects_write,  CR, FRIC);
				hipMemcpy(dev_objects, dev_objects_write, size, hipMemcpyDeviceToDevice);
				step << <numBlocks, blockSize >> > (N, dev_objects, dev_objects_write, dt);
				hipMemcpy(dev_objects, dev_objects_write, size, hipMemcpyDeviceToDevice);
				gravitate << <numBlocks, blockSize >> > (N, dev_objects, dev_objects_write, G, dt);

				hipMemcpy(objects, dev_objects_write, size, hipMemcpyDeviceToHost);
				hipDeviceSynchronize();

				// Host Gravitate
				/*
				for (int i = 0; i < N; i++) {
					for (int j = 0; j < N; j++) {
						PhysObj& obj1 = objects[i];
						PhysObj& obj2 = objects[j];
						if (i!=j) {
							obj1.vel += normalize(obj1.pos, obj2.pos) * (G * dt * obj2.mass / (pow(dist(obj1.pos, obj2.pos), 2)));
							obj2.vel += normalize(obj2.pos, obj1.pos) * (G * dt * obj1.mass / (pow(dist(obj1.pos, obj2.pos), 2)));
						}
					}
				}
				*/
				
				if (frame_tick >= tpf) {
					main_loop();
					frame_tick = 0;
				}
				frame_tick++;
				

			}
#endif
		}
	}

	//Free resources and close SDL
	close();
	free(objects);
	return 0;
}
// Run program: Ctrl + F5 or Debug > Start Without Debugging menu
// Debug program: F5 or Debug > Start Debugging menu

// Tips for Getting Started: 
//   1. Use the Solution Explorer window to add/manage files
//   2. Use the Team Explorer window to connect to source control
//   3. Use the Output window to see build output and other messages
//   4. Use the Error List window to view errors
//   5. Go to Project > Add New Item to create new code files, or Project > Add Existing Item to add existing code files to the project
//   6. In the future, to open this project again, go to File > Open > Project and select the .sln file
